
#include <hip/hip_runtime.h>
#ifdef PRECICE_WITH_CUDA

#include <cublas_v2.h>
#include <cuda.h>
#include <cuda_runtime.h>
#include <cuda_runtime_api.h>
#include <cusolverDn.h>
#include <ginkgo/ginkgo.hpp>
#include "device_launch_parameters.h"
#include "mapping/device/CudaQRSolver.cuh"
#include "profiling/Event.hpp"
#include "profiling/EventUtils.hpp"

void computeQRDecompositionCuda(const std::shared_ptr<gko::Executor> &exec, GinkgoMatrix *A_Q, GinkgoVector *R)
{
  auto scope_guard = exec->get_scoped_device_id_guard();

  void *dWork{};
  int * devInfo{};

  // Allocating important CUDA variables
  cudaError_t cudaErrorCode = cudaMalloc((void **) &dWork, sizeof(double));
  assert(cudaSuccess == cudaErrorCode);
  cudaErrorCode = cudaMalloc((void **) &devInfo, sizeof(int));
  assert(cudaSuccess == cudaErrorCode);

  cusolverDnHandle_t solverHandle;
  cusolverDnCreate(&solverHandle);
  // NOTE: It's important to transpose since cuSolver assumes column-major memory layout
  // Making a copy since every value will be overridden
  auto A_T = gko::share(gko::matrix::Dense<>::create(exec, gko::dim<2>(A_Q->get_size()[1], A_Q->get_size()[0])));
  A_Q->transpose(gko::lend(A_T));

  // Setting dimensions for solver
  const unsigned int M = A_T->get_size()[1];
  const unsigned int N = A_T->get_size()[0];

  const int lda = max(1, M);
  const int k   = min(M, N);

  size_t dLwork_geqrf = 0;
  size_t dLwork_orgqr = 0;
  size_t dLwork       = 0;

  size_t hLwork_geqrf = 0;
  size_t hLwork       = 0;

  double *dTau{};
  cudaErrorCode = cudaMalloc((void **) &dTau, sizeof(double) * M);
  assert(cudaSuccess == cudaErrorCode);

  precice::profiling::Event calculateQRDecompEvent{"calculateQRDecomp"};

  // Query working space of geqrf and orgqr
  cusolverStatus_t cusolverStatus = cusolverDnXgeqrf_bufferSize(solverHandle, nullptr, M, N, CUDA_R_64F, A_T->get_values(), lda, CUDA_R_64F, dTau, CUDA_R_64F, &dLwork_geqrf, &hLwork_geqrf);
  // PRECICE_ASSERTs collide with cuda for some (non-extensively investigated) reason
  assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);
  cusolverStatus = cusolverDnDorgqr_bufferSize(solverHandle, M, N, k, A_T->get_values(), lda, dTau, (int *) &dLwork_orgqr);
  assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);
  dLwork                    = (dLwork_geqrf > dLwork_orgqr) ? dLwork_geqrf : dLwork_orgqr;
  cudaErrorCode = cudaMalloc((void **) &dWork, sizeof(double) * dLwork);
  assert(cudaSuccess == cudaErrorCode);

  void *hWork{};
  // Compute QR factorization
  cusolverStatus = cusolverDnXgeqrf(solverHandle, nullptr, M, N, CUDA_R_64F, A_T->get_values(), lda, CUDA_R_64F, dTau, CUDA_R_64F, dWork, dLwork, hWork, hLwork, devInfo);
  cudaErrorCode  = cudaDeviceSynchronize();
  assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);
  assert(cudaSuccess == cudaErrorCode);

  // Copy A_T to R s.t. the upper triangle corresponds to R
  A_T->transpose(gko::lend(R));

  // Compute Q
  cusolverStatus = cusolverDnDorgqr(solverHandle, M, N, k, A_T->get_values(), lda, dTau, (double *) dWork, dLwork, devInfo);
  cudaErrorCode  = cudaDeviceSynchronize();
  assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);
  assert(cudaSuccess == cudaErrorCode);

  A_T->transpose(gko::lend(A_Q));

  cudaErrorCode = cudaDeviceSynchronize();
  assert(cudaSuccess == cudaErrorCode);
  calculateQRDecompEvent.stop();

  // Free the utilized memory
  cudaErrorCode = cudaFree(dTau);
  assert(cudaSuccess == cudaErrorCode);
  cudaErrorCode = cudaFree(dWork);
  assert(cudaSuccess == cudaErrorCode);
  cudaErrorCode = cudaFree(devInfo);
  assert(cudaSuccess == cudaErrorCode);
  cusolverStatus = cusolverDnDestroy(solverHandle);
  assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);
}

void solvewithQRDecompositionCuda(const std::shared_ptr<gko::Executor> &exec, GinkgoMatrix *U, GinkgoVector *x, GinkgoVector *rhs, GinkgoMatrix *matQ, GinkgoVector *in_vec)
{
  auto scope_guard = exec->get_scoped_device_id_guard();

  cublasHandle_t handle;
  cublasStatus_t cublasStatus = cublasCreate(&handle);
  assert(cublasStatus == CUBLAS_STATUS_SUCCESS);
  double a     = 1;
  double b     = 0;
  cublasStatus = cublasDgemv(handle, CUBLAS_OP_T,
                             matQ->get_size()[0], matQ->get_size()[1],
                             &a,
                             matQ->get_values(), matQ->get_size()[0],
                             in_vec->get_values(), 1,
                             &b,
                             rhs->get_values(), 1);
  assert(cublasStatus == CUBLAS_STATUS_SUCCESS);

  cublasFillMode_t  uplo  = CUBLAS_FILL_MODE_LOWER;
  cublasOperation_t trans = CUBLAS_OP_T;

  // unit triangular = diag = 1
  cublasDiagType_t diag    = CUBLAS_DIAG_NON_UNIT;
  int              rows    = rhs->get_size()[0];
  const int        lda     = max(1, rows);

  cublasStatus = cublasDtrsv(handle, uplo,
                             trans, diag,
                             rows, U->get_values(), lda,
                             rhs->get_values(), 1);
  assert(cublasStatus == CUBLAS_STATUS_SUCCESS);

  // In case we refactor the code in the future to make use of
  // dtrsm instead of dtrsv (processing vector data as a whole),
  // the following holds
  // double           alpha   = 1.0;
  // int              columns = 1;
  // const int        ldb     = max(1, rows);
  // cublasSideMode_t  side  = CUBLAS_SIDE_LEFT;
  // cublasStatus = cublasDtrsm(handle,
  //                            side,
  //                            uplo,
  //                            trans,
  //                            diag,
  //                            rows,
  //                            columns,
  //                            &alpha,
  //                            U->get_values(), lda,
  //                            rhs->get_values(),
  //                            ldb);

  cudaDeviceSynchronize();
  *x           = *rhs;
  cublasStatus = cublasDestroy(handle);
  assert(cublasStatus == CUBLAS_STATUS_SUCCESS);
}
#endif
