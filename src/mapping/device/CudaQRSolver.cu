
#include <hip/hip_runtime.h>
#ifdef PRECICE_WITH_CUDA

#include <cublas_v2.h>
#include <cuda.h>
#include <cuda_runtime.h>
#include <cuda_runtime_api.h>
#include <cusolverDn.h>
#include <ginkgo/ginkgo.hpp>
#include "device_launch_parameters.h"
#include "mapping/device/CudaQRSolver.cuh"
#include "profiling/Event.hpp"
#include "profiling/EventUtils.hpp"

void computeQRDecompositionCuda(const std::shared_ptr<gko::Executor> &exec, GinkgoMatrix *A_Q, GinkgoVector *R)
{
  auto scope_guard = exec->get_scoped_device_id_guard();

  void *dWork{};
  int * devInfo{};

  // Allocating important CUDA variables
  cudaMalloc((void **) &dWork, sizeof(double));
  cudaMalloc((void **) &devInfo, sizeof(int));

  cusolverDnHandle_t solverHandle;
  cusolverDnCreate(&solverHandle);
  // NOTE: It's important to transpose since cuSolver assumes column-major memory layout
  // Making a copy since every value will be overridden
  auto A_T = gko::share(gko::matrix::Dense<>::create(exec, gko::dim<2>(A_Q->get_size()[1], A_Q->get_size()[0])));
  A_Q->transpose(gko::lend(A_T));

  // Setting dimensions for solver
  const unsigned int M = A_T->get_size()[1];
  const unsigned int N = A_T->get_size()[0];

  const int lda = max(1, M);
  const int k   = min(M, N);

  size_t dLwork_geqrf = 0;
  size_t dLwork_orgqr = 0;
  size_t dLwork       = 0;

  size_t hLwork_geqrf = 0;
  size_t hLwork       = 0;

  double *dTau{};
  cudaMalloc((void **) &dTau, sizeof(double) * M);

  precice::profiling::Event calculateQRDecompEvent{"calculateQRDecomp"};

  // Query working space of geqrf and orgqr
  cusolverStatus_t cusolverStatus = cusolverDnXgeqrf_bufferSize(solverHandle, nullptr, M, N, CUDA_R_64F, A_T->get_values(), lda, CUDA_R_64F, dTau, CUDA_R_64F, &dLwork_geqrf, &hLwork_geqrf);
  // PRECICE_ASSERTs collide with cuda for some (non-extensively investigated) reason
  assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);
  cusolverStatus = cusolverDnDorgqr_bufferSize(solverHandle, M, N, k, A_T->get_values(), lda, dTau, (int *) &dLwork_orgqr);
  assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);
  dLwork                    = (dLwork_geqrf > dLwork_orgqr) ? dLwork_geqrf : dLwork_orgqr;
  cudaError_t cudaErrorCode = cudaMalloc((void **) &dWork, sizeof(double) * dLwork);
  assert(cudaSuccess == cudaErrorCode);

  void *hWork{};
  // Compute QR factorization
  cusolverStatus = cusolverDnXgeqrf(solverHandle, nullptr, M, N, CUDA_R_64F, A_T->get_values(), lda, CUDA_R_64F, dTau, CUDA_R_64F, dWork, dLwork, hWork, hLwork, devInfo);
  cudaErrorCode  = cudaDeviceSynchronize();
  assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);
  assert(cudaSuccess == cudaErrorCode);

  // Copy A_T to R s.t. the upper triangle corresponds to R
  A_T->transpose(gko::lend(R));

  // Compute Q
  cusolverStatus = cusolverDnDorgqr(solverHandle, M, N, k, A_T->get_values(), lda, dTau, (double *) dWork, dLwork, devInfo);
  cudaErrorCode  = cudaDeviceSynchronize();
  assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);
  assert(cudaSuccess == cudaErrorCode);

  A_T->transpose(gko::lend(A_Q));

  cudaDeviceSynchronize();
  calculateQRDecompEvent.stop();

  // Free the utilized memory
  cudaFree(dTau);
  cudaFree(dWork);
  cudaFree(devInfo);
  cusolverDnDestroy(solverHandle);
}
#endif
