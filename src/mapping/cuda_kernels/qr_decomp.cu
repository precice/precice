
#include <hip/hip_runtime.h>
#ifdef PRECICE_WITH_CUDA

#include <cublas_v2.h>
#include <cuda.h>
#include <cuda_runtime.h>
#include <cuda_runtime_api.h>
#include <cusolverDn.h>
#include <ginkgo/ginkgo.hpp>
#include "device_launch_parameters.h"
#include "mapping/cuda_kernels/qr_decomp.cuh"
#include "utils/Event.hpp"
#include "utils/EventUtils.hpp"

QRSolver::QRSolver(const int deviceId)
{
  cudaGetDevice(&cudaBackupDeviceId);
  cudaSetDevice(deviceId);

  // Allocating important CUDA variables
  cudaMalloc((void **) &dWork, sizeof(double));
  cudaMalloc((void **) &devInfo, sizeof(int));

  cusolverDnCreate(&solverHandle);
}

QRSolver::~QRSolver()
{
  // Freeing CUDA variables
  cudaFree(dTau);
  cudaFree(dWork);
  cudaFree(devInfo);
  cusolverDnDestroy(solverHandle);
}

void QRSolver::computeQR(const std::shared_ptr<gko::Executor> &exec, GinkgoMatrix *A_Q, GinkgoMatrix *R)
{
  // NOTE: It's important to transpose since cuSolver assumes column-major memory layout
  // Making a copy since every value will be overridden
  auto A_T = gko::share(GinkgoMatrix::create(exec, gko::dim<2>(A_Q->get_size()[1], A_Q->get_size()[0])));
  A_Q->transpose(gko::lend(A_T));

  // Setting dimensions for solver
  const unsigned int M = A_T->get_size()[1];
  const unsigned int N = A_T->get_size()[0];

  const int lda = max(1, M);
  const int k   = min(M, N);

  size_t dLwork_geqrf = 0;
  size_t dLwork_orgqr = 0;
  size_t dLwork       = 0;

  size_t hLwork_geqrf = 0;
  size_t hLwork       = 0;

  cudaMalloc((void **) &dTau, sizeof(double) * M);

  precice::utils::Event calculateQRDecompEvent{"calculateQRDecomp"};

  // Query working space of geqrf and orgqr
  cusolverStatus = cusolverDnXgeqrf_bufferSize(solverHandle, nullptr, M, N, CUDA_R_64F, A_T->get_values(), lda, CUDA_R_64F, dTau, CUDA_R_64F, &dLwork_geqrf, &hLwork_geqrf);
  // PRECICE_ASSERTs collide with cuda for some (non-extensively investigated) reason
  assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);
  cusolverStatus = cusolverDnDorgqr_bufferSize(solverHandle, M, N, k, A_T->get_values(), lda, dTau, (int *) &dLwork_orgqr);
  assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);
  dLwork        = (dLwork_geqrf > dLwork_orgqr) ? dLwork_geqrf : dLwork_orgqr;
  cudaErrorCode = cudaMalloc((void **) &dWork, sizeof(double) * dLwork);
  assert(cudaSuccess == cudaErrorCode);

  // Compute QR factorization
  cusolverStatus = cusolverDnXgeqrf(solverHandle, nullptr, M, N, CUDA_R_64F, A_T->get_values(), lda, CUDA_R_64F, dTau, CUDA_R_64F, dWork, dLwork, hWork, hLwork, devInfo);
  cudaErrorCode  = cudaDeviceSynchronize();
  assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);
  assert(cudaSuccess == cudaErrorCode);

  // Copy A_T to R s.t. the upper triangle corresponds to R
  A_T->transpose(gko::lend(R));

  // Compute Q
  cusolverStatus = cusolverDnDorgqr(solverHandle, M, N, k, A_T->get_values(), lda, dTau, (double *) dWork, dLwork, devInfo);
  cudaErrorCode  = cudaDeviceSynchronize();
  assert(cusolverStatus == CUSOLVER_STATUS_SUCCESS);
  assert(cudaSuccess == cudaErrorCode);

  A_T->transpose(gko::lend(A_Q));

  cudaDeviceSynchronize();

  calculateQRDecompEvent.stop();

  cudaSetDevice(cudaBackupDeviceId); // Switch back to the GPU used for all coupled solvers
}
#endif
